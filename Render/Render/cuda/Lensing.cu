#include "hip/hip_runtime.h"
#include "Lensing.cuh"

#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <sstream>
#include <iterator>

#include <GL\glew.h>

#include <FreeImage/FreeImage.h>

namespace ginkgo
{

#define DATADEBUG 1

#define ZEROPADDING 0

	std::vector<std::vector<glm::dvec4>> lensing()
	{
		int N =
#if ZEROPADDING
			2 *
#endif
		18; // ex: N = 2*32 //future: 128 X 128, N = 128 // N is the sidelength of the image

		int block_size_x = 32;
		int block_size_y = 32;

		double *kx, *ky, *r;
		kx = (double *)malloc(sizeof(double) * N);
		ky = (double *)malloc(sizeof(double) * N);
		r = (double *)malloc(sizeof(double) * N * N);

		double *kx_d, *ky_d, *r_d;
		hipfftDoubleComplex *r_complex_d;
		hipMalloc((void **)&kx_d, sizeof(double) * N);
		hipMalloc((void **)&ky_d, sizeof(double) * N);
		hipMalloc((void **)&r_d, sizeof(double) * N * N);
		hipMalloc((void **)&r_complex_d, sizeof(hipfftDoubleComplex) * N * N);

		for (int y = 0; y < N; y++)
			for (int x = 0; x < N; x++)
				r[x + y * N] = sin(
					exp(-((x - N / 2.0f) * (x - N / 2.0f) + (N / 2.0f - y) * (N / 2.0f - y)) / (50 * 50))
				);

#if ZEROPADDING
		for (int y = 0; y < N; y++)
			for (int x = 0; x < N; x++)
				if (x < N / 4.0 || x > N * 3.0 / 4.0 || y < N / 4.0 || y > N*3.0 / 4.0)
					r[x + y*N] = 0;
#endif

		double* r_inital = (double *)malloc(sizeof(double) * N * N);
		for (int i = 0; i < N * N; i++)
			r_inital[i] = r[i];

		for (int i = 0; i < N; i++)
		{
			kx[i] = i - N / 2.0f; //centers kx values to be at center of image
			ky[i] = N / 2.0f - i; //centers ky values to be at center of image
		}

#if DATADEBUG
		writeDoubleArrayToBitmapImage("DataDebug/SpatialDensity.bmp", N, r);
		write("DataDebug/SpatialDensity.csv", r, N, N);
#endif
		hipMemcpy(kx_d, kx, sizeof(double) * N, hipMemcpyHostToDevice);
		hipMemcpy(ky_d, ky, sizeof(double) * N, hipMemcpyHostToDevice);
		hipMemcpy(r_d, r, sizeof(double) * N * N, hipMemcpyHostToDevice);

		hipfftHandle plan;
		hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

		/* Compute the execution configuration
		NB: block_size_x*block_size_y = number of threads */
		dim3 dimBlock(block_size_x, block_size_y);
		dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
		/* Handle N not multiple of block_size_x or block_size_y */
		if (N % block_size_x != 0) dimGrid.x += 1;
		if (N % block_size_y != 0) dimGrid.y += 1;

		real2complex << < dimGrid, dimBlock >> > (r_complex_d, r_d, N);

		hipfftExecZ2Z(plan, r_complex_d, r_complex_d, HIPFFT_FORWARD);
		//solve_poisson << <dimGrid, dimBlock >> > (r_complex_d, kx_d, ky_d, N);
		hipfftExecZ2Z(plan, r_complex_d, r_complex_d, HIPFFT_BACKWARD);

		double scale = 1.0f / (N * N);// *2E3;
		complex2real_scaled << <dimGrid, dimBlock >> > (r_d, r_complex_d, scale, N);

		hipMemcpy(r, r_d, sizeof(double) * N * N, hipMemcpyDeviceToHost);

#if DATADEBUG
		writeDoubleArrayToBitmapImage("DataDebug/GravitationalPotential.bmp", N, r);
		write("DataDebug/GravitationalPotential.csv", r, N, N);
#endif

		std::vector<std::vector<glm::dvec4>> normals = generateNormals(r, N, N);
		glm::dvec4 normal = normals[N / 2][N / 2];
		std::cout << "Center: " << ":\t" << normal.x << " " << normal.y << " " << normal.z << std::endl;
		//normals[center] = glm::dvec3(0.0f, 0.0f, 0.0f);
#if DATADEBUG
		writeNormalsToBitmapImage("DataDebug/Normals.bmp", N - 2, normals);
		writeNormals("DataDebug/Normals.csv", normals);
#endif
		//system("pause");

		/* Destroy plan and clean up memory on device*/
		free(kx);
		free(ky);
		free(r);
		free(r_inital);
		hipfftDestroy(plan);
		hipFree(r_complex_d);
		hipFree(kx_d);

		std::cout << "Normal Size: " << normals.size() << "\n";

		return normals;
	}

	__global__ void real2complex(hipfftDoubleComplex *c, double *a, int N)
	{
		/* compute idx and idy, the location of the element in the original NxN array */
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idx < N && idy < N)
		{
			int index = idx + idy * N;
			c[index].x = a[index];
			c[index].y = 0.0f;
		}
	}

	__global__ void solve_poisson(hipfftDoubleComplex *c, double *kx, double *ky, int N)
	{
		/* compute idx and idy, the location of the element in the original NxN array */
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idx < N && idy < N)
		{
			int index = idx + idy * N;
			double scale = -(kx[idx] * kx[idx] + ky[idy] * ky[idy]);
			if (idx == N / 2 && idy == N / 2) scale = -1.0f;
			scale = 1.0f / scale;
			c[index].x *= scale;
			c[index].y *= scale;
		}
	}

	__global__ void complex2real_scaled(double *a, hipfftDoubleComplex *c, double scale, int N)
	{
		/* compute idx and idy, the location of the element in the original NxN array */
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idx < N && idy < N)
		{
			int index = idx + idy * N;
			a[index] = scale * c[index].x;
		}
	}

	void writeFast(const std::string& path, double* data, int NxN, int mode)
	{
		std::ofstream out(path.c_str(), mode);

		for (int i = 0; i < NxN; i++)
			out.write(reinterpret_cast<char *>(&data[i]), sizeof(double));

		out.close();
	}
	void readFast(const std::string& path, double* data, int NxN, int mode)
	{
		std::ifstream in(path.c_str(), mode);

		for (int i = 0; i < NxN; i++)
			in.read(reinterpret_cast<char *>(&data[i]), sizeof(double));

		in.close();
	}
	void write(const std::string& path, double* data, int height, int width, int mode)
	{
		std::ofstream fout(path, mode);

		for (int y = 0; y < height; y++)
		{
			for (int x = 0; x < width; x++)
				fout << data[x + y * width] << ",";
			fout << std::endl;
		}
		fout.close();
	}
	void write(const std::string& path, std::vector<double>& data, int height, int width, int mode)
	{
		std::ofstream fout(path, mode);

		for (int y = 0; y < height; y++)
		{
			for (int x = 0; x < width; x++)
				fout << data[x + y * width] << ",";
			fout << std::endl;
		}
		fout.close();
	}
	void writeCFast(const std::string& filename, const double* data, size_t size)
	{
		FILE* pFile;
		pFile = fopen(filename.c_str(), "wb");
		for (unsigned long long j = 0; j < 1024; ++j) {
			//Some calculations to fill a[]
			fwrite(data, 1, size * sizeof(double), pFile);
		}
		fclose(pFile);
	}
	double* readCFast(const std::string& filename, size_t size)
	{
		FILE* pFile;
		long lSize;
		double* data;
		size_t result;

		pFile = fopen(filename.c_str(), "rb");
		if (pFile == NULL) { fputs("File error in readBOOMFast()", stderr); system("pause"); }

		// obtain file size:
		fseek(pFile, 0, SEEK_END);
		lSize = ftell(pFile);
		rewind(pFile);

		// allocate memory to contain the whole file:
		data = (double*)malloc(sizeof(char)*lSize);
		if (data == NULL) { fputs("Memory error in readBOOMFast()", stderr); system("pause"); }

		// copy the file into the buffer:
		result = fread(data, 1, lSize, pFile);
		if (result != lSize) { fputs("Reading error in readBOOMFast()", stderr); system("pause"); }

		/* the whole file is now loaded in the memory buffer. */

		// terminate
		fclose(pFile);
		return data; //remember to free(data)
	}
	void writeDoubleArrayToBitmapImage(const char* filename, int N, double *r)
	{
		double max = r[0];
		double min = r[0];
		for (int i = 0; i < N * N; i++)
		{
			if (r[i] > max)
				max = r[i];
			if (r[i] < min)
				min = r[i];
		}

		unsigned char* pixels = new unsigned char[3 * N *N];
		for (int i = 0; i < 3 * N *N; i += 3)
			//for (int i = 3 * (width * height - 1); i >= 0 ; i -= 3)
		{
			int greyColor = static_cast<int>(255.0 / (max - min) * (r[i / 3] - min));
			pixels[i + 2] = greyColor; //red
			pixels[i + 1] = greyColor; //green
			pixels[i + 0] = greyColor; //blue
		}
		FIBITMAP* image = FreeImage_ConvertFromRawBits(pixels, N, N, 3 * N, 24, 0xFF0000, 0x00FF00, 0x0000FF, false);
		FreeImage_FlipVertical(image);
		FreeImage_Save(FIF_BMP, image, filename, 0);
		FreeImage_Unload(image);
		delete[] pixels;
	}
	void writeNormalsToBitmapImage(const char* filename, int N, const std::vector<std::vector<glm::dvec4>>& normals)
	{
		glm::dvec4 max = normals[0][0];
		glm::dvec4 min = normals[0][0];
		for (int r = 0; r < N; r++)
		{
			for (int c = 0; c < N; c++)
			{

				if (normals[r][c].x > max.x)
					max.x = normals[r][c].x;
				if (normals[r][c].y > max.y)
					max.y = normals[r][c].y;
				if (normals[r][c].z > max.z)
					max.z = normals[r][c].z;

				if (normals[r][c].x < min.x)
					min.x = normals[r][c].x;
				if (normals[r][c].y < min.y)
					min.y = normals[r][c].y;
				if (normals[r][c].z < min.z)
					min.z = normals[r][c].z;
			}
		}

		unsigned char* pixels = new unsigned char[3 * N * N];
		for (int r = 0; r < N; r++)
		{
			for (int c = 0; c < N; c++)
			{

				glm::dvec4 normal = normals[r][c];
				//255.0 / (max - min) * (r[i / 3] - min)

				pixels[r*N + c + 2] = static_cast<int>((255.0 * (0.50 * (normal.x + 1))) + 0.50); //red  - x direction
				pixels[r*N + c + 1] = static_cast<int>((255.0 * (0.50 * (normal.y + 1))) + 0.50); //green - y direction
				pixels[r*N + c + 0] = static_cast<int>((255.0 * (0.50 * (normal.z + 1))) + 0.50); //blue - z direction
			}
		}
		FIBITMAP* image = FreeImage_ConvertFromRawBits(pixels, N, N, 3 * N, 24, 0xFF0000, 0x00FF00, 0x0000FF, false);
		FreeImage_Save(FIF_BMP, image, filename, 0);
		FreeImage_Unload(image);
		delete[] pixels;
	}
	void writeNormals(const std::string& path, std::vector<std::vector<glm::dvec4>>& data, int mode)
	{
		std::ofstream fout(path, mode);

		int height = data.size();
		int width = data[0].size();

		for (int y = 0; y < height; y++)
		{
			for (int x = 0; x < width; x++)
			{
				fout << "\"=\"\"" << data[y][x].x << ":" << data[y][x].y << ":" << data[y][x].z << "\"\"\"" << ",";
			}
			fout << std::endl;
		}
		fout.close();
	}
	std::vector<double> const read(const std::string& path, int NxN)
	{
		std::ifstream stream(path);
		std::string lineOut;
		std::vector<double> data;
		while (std::getline(stream, lineOut))
		{
			int start = 0;
			int nextComma = 0;

			while ((nextComma = lineOut.find(',', start)) != std::string::npos)
			{
				std::string val = lineOut.substr(start, nextComma - start);
				start = nextComma + 1;
				data.emplace_back(std::stod(val));
			}
		}

		return data;
	}
	std::vector<std::vector<glm::dvec4>> generateNormals(double* data, int height, int width)
	{
		std::vector<std::vector<glm::dvec4>> normals;
		int normal_size = (width - 2) * (height - 2);
		std::vector<double> dfx;
		std::vector<double> dfy;

		for (int y = 1; y < height - 1; y++)
		{
			for (int x = 1; x < width - 1; x++)
			{
				dfx.emplace_back((data[(x + 1) + y * height] - data[(x - 1) + y * height]) / 2.0f);
				dfy.emplace_back((data[x + (y + 1) * height] - data[x + (y - 1) * height]) / 2.0f);
			}
		}

		double dfx_max = dfx[0];
		double dfy_max = dfy[0];
		for (int i = 0; i < normal_size; i++)
		{
			if (dfx[i] > dfx_max)
				dfx_max = dfx[i];
			if (dfy[i] > dfy_max)
				dfy_max = dfy[i];
		}

		dfx_max = abs(dfx_max);
		dfy_max = abs(dfy_max);

		double max = (dfx_max > dfy_max) ? dfx_max : dfy_max;
		double a = 0.99 / max; //double a = 0.1/max; TODO edit

		for (int r = 0; r < height - 2; r++)
		{
			std::vector<glm::dvec4> row;
			normals.emplace_back(row);
			for (int c = 0; c < width - 2; c++)
			{
				double nx = a * dfx[r*(height - 2) + c];
				double ny = a * dfy[r*(height - 2) + c];
				double nz = sqrt(1.0 - nx * nx - ny * ny);
				normals[r].emplace_back(glm::dvec4(nx, ny, nz, 1.0f));
			}
		}

		return normals;
	}

}



/*
TODOs:
http://gpgpu.org/static/sc2007/SC07_CUDA_3_Libraries.pdf

convert b/w image to spatial density

writetoimage values -> understand why
colorings is wrong -> understand why

zero padding is wrong

remove static_const<int> thing -> nah its fine

put solution on stackoverflow -> yup

read and write to data file -> yup
read and write scaled data to bitmap

make more efficient using shared memory, etc.

fix up pading currectly, so it adds it onto the image, instead of changing the existing data values of the original image

do const stuff

even faster read write methods
*/